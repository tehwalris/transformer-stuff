#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include <stdio.h>
#include <chrono>
#include <cassert>

#define CUDA_CHECK(err)                                                         \
  do                                                                            \
  {                                                                             \
    hipError_t err_ = (err);                                                   \
    if (err_ != hipSuccess)                                                    \
    {                                                                           \
      fprintf(stderr, "CUDA error %d at %s:%d: %s\n", err_, __FILE__, __LINE__, \
              hipGetErrorString(err_));                                        \
      exit(1);                                                                  \
    }                                                                           \
  } while (0)

void fill_rand_char4(int n, char4 *arr)
{
  union int_char4
  {
    int i;
    char4 c;
  };

  assert(n % 4 == 0);
  for (int i = 0; i < n; i += 4)
  {
    int_char4 temp;
    temp.i = rand();
    // HACK because rand() only generates non-negative numbers
    temp.i *= temp.i & 1 ? 1 : -1;
    arr[i / 4] = temp.c;
  }
}

float rand_float_neg_1_1()
{
  return (float)rand() / (float)RAND_MAX * 2.0f - 1.0f;
}

void init_cpu(int n, char4 *A, float *A_scale, char4 *x, float *x_scale, float *y, char4 *y_quantized, float *y_scale, float *z)
{
  fill_rand_char4(n * n, A);
  for (int i = 0; i < n; i++)
  {
    A_scale[i] = rand_float_neg_1_1();
  }
  fill_rand_char4(n, x);
  *x_scale = rand_float_neg_1_1();
  for (int i = 0; i < n; i++)
  {
    y[i] = 0.0f;
  }
  for (int i = 0; i < n; i += 4)
  {
    y_quantized[i / 4].x = 0;
    y_quantized[i / 4].y = 0;
    y_quantized[i / 4].z = 0;
    y_quantized[i / 4].w = 0;
  }
  *y_scale = 0.0f;
  for (int i = 0; i < n; i++)
  {
    z[i] = 0.0f;
  }
}

void clear_cpu(int n, float *y, char4 *y_quantized, float *y_scale, float *z)
{
  for (int i = 0; i < n; i++)
  {
    y[i] = 0.0f;
  }
  for (int i = 0; i < n; i += 4)
  {
    y_quantized[i / 4].x = 0;
    y_quantized[i / 4].y = 0;
    y_quantized[i / 4].z = 0;
    y_quantized[i / 4].w = 0;
  }
  *y_scale = 0.0f;
  for (int i = 0; i < n; i++)
  {
    z[i] = 0.0f;
  }
}

__global__ void mul_gpu(int n, char4 const *__restrict__ A, char4 const *__restrict__ x, float *__restrict__ y)
{
  for (int i_row = blockIdx.y * blockDim.y + threadIdx.y; i_row < n; i_row += blockDim.y * gridDim.y)
  {
    int sum = 0;
    for (int i_col = threadIdx.x; i_col < n / 4; i_col += blockDim.x)
    {
      sum = __dp4a(A[(i_row * n) / 4 + i_col], x[i_col], sum);
    }
    atomicAdd(&y[i_row], float(sum));
  }
}

void mul_cpu(int n, char4 *A, char4 *x, float *y)
{
  for (int i = 0; i < n; i++)
  {
    int sum = 0;
    for (int j = 0; j < n; j += 4)
    {
      sum += int(A[(i * n + j) / 4].x) * int(x[j / 4].x);
      sum += int(A[(i * n + j) / 4].y) * int(x[j / 4].y);
      sum += int(A[(i * n + j) / 4].z) * int(x[j / 4].z);
      sum += int(A[(i * n + j) / 4].w) * int(x[j / 4].w);
    }
    y[i] = float(sum);
  }
}

__global__ void post_mul_scale_gpu(int n, float *y, float *A_scale, float x_scale)
{
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x)
  {
    y[i] *= A_scale[i] * x_scale;
  }
}

void post_mul_scale_cpu(int n, float *y, float *A_scale, float x_scale)
{
  for (int i = 0; i < n; i++)
  {
    y[i] *= A_scale[i] * x_scale;
  }
}

void quantize_q8_cpu(int n, float *input, char4 *output, float *unquantize_scale)
{
  float abs_max = 0.0f;
  for (int i = 0; i < n; i++)
  {
    abs_max = std::max(abs_max, std::abs(input[i]));
  }

  *unquantize_scale = abs_max * (1.0f / 127.0f);
  float quantize_scale = 1.0f / *unquantize_scale;

  for (int i = 0; i < n; i += 4)
  {
    output[i / 4].x = int8_t(std::round(input[i + 0] * quantize_scale));
    output[i / 4].y = int8_t(std::round(input[i + 1] * quantize_scale));
    output[i / 4].z = int8_t(std::round(input[i + 2] * quantize_scale));
    output[i / 4].w = int8_t(std::round(input[i + 3] * quantize_scale));
  }
}

int main(void)
{
  srand(0);

  int N = 4096;
  uint32_t num_iterations = 10;

  char4 *A;
  float *A_scale;
  char4 *x;
  float x_scale;
  float *y;
  char4 *y_quantized;
  float y_scale;
  float *z;
  assert(N % 4 == 0);
  CUDA_CHECK(hipMallocManaged(&A, N * N / 4 * sizeof(char4)));
  CUDA_CHECK(hipMallocManaged(&A_scale, N * sizeof(float)));
  CUDA_CHECK(hipMallocManaged(&x, N / 4 * sizeof(char4)));
  CUDA_CHECK(hipMallocManaged(&y, N * sizeof(float)));
  CUDA_CHECK(hipMallocManaged(&y_quantized, N / 4 * sizeof(char4)));
  CUDA_CHECK(hipMallocManaged(&z, N * sizeof(float)));

  init_cpu(N, A, A_scale, x, &x_scale, y, y_quantized, &y_scale, z);

  // GPU version
  {
    clear_cpu(N, y, y_quantized, &y_scale, z);

    assert(N % 4 == 0);
    dim3 block_size_mul(32, 8);
    dim3 grid_size_mul(1, (N + block_size_mul.y - 1) / block_size_mul.y);

    int block_size_scale(256);
    int grid_size_scale((N + block_size_scale - 1) / block_size_scale);

    mul_gpu<<<grid_size_mul, block_size_mul>>>(N, A, x, y);
    post_mul_scale_gpu<<<grid_size_scale, block_size_scale>>>(N, y, A_scale, x_scale);
    CUDA_CHECK(hipDeviceSynchronize());
    quantize_q8_cpu(N, y, y_quantized, &y_scale); // TODO GPU
    mul_gpu<<<grid_size_mul, block_size_mul>>>(N, A, y_quantized, z);
    post_mul_scale_gpu<<<grid_size_scale, block_size_scale>>>(N, z, A_scale, y_scale);
    CUDA_CHECK(hipDeviceSynchronize());

    float sum_y = 0.0f;
    float sum_z = 0.0f;
    for (int i = 0; i < N; i++)
    {
      sum_y += y[i];
      sum_z += z[i];
    }
    std::cout << "GPU result: y[0] = " << y[0] << ", sum(y) = " << sum_y << ", z[0] = " << z[0] << ", sum(z) = " << sum_z << std::endl;

    CUDA_CHECK(hipMemPrefetchAsync(A, N * N / 4 * sizeof(char4), 0));
    CUDA_CHECK(hipMemPrefetchAsync(x, N / 4 * sizeof(char4), 0));
    CUDA_CHECK(hipMemPrefetchAsync(y, N * sizeof(float), 0));
    // TODO

    printf("Starting GPU benchmark\n");
    auto start = std::chrono::high_resolution_clock::now();
    for (uint32_t i = 0; i < num_iterations; i++)
    {
      mul_gpu<<<grid_size_mul, block_size_mul>>>(N, A, x, y);
      post_mul_scale_gpu<<<grid_size_scale, block_size_scale>>>(N, y, A_scale, x_scale);
      CUDA_CHECK(hipDeviceSynchronize());
      quantize_q8_cpu(N, y, y_quantized, &y_scale); // TODO GPU
      mul_gpu<<<grid_size_mul, block_size_mul>>>(N, A, y_quantized, z);
      post_mul_scale_gpu<<<grid_size_scale, block_size_scale>>>(N, z, A_scale, y_scale);
    }
    CUDA_CHECK(hipDeviceSynchronize());
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed = end - start;
    printf("%f ms per iteration\n", elapsed.count() / num_iterations * 1e3f);
  }

  // CPU version
  {
    clear_cpu(N, y, y_quantized, &y_scale, z);

    mul_cpu(N, A, x, y);
    post_mul_scale_cpu(N, y, A_scale, x_scale);
    quantize_q8_cpu(N, y, y_quantized, &y_scale);
    mul_cpu(N, A, y_quantized, z);
    post_mul_scale_cpu(N, z, A_scale, y_scale);

    CUDA_CHECK(hipMemPrefetchAsync(A, N * N / 4 * sizeof(char4), hipCpuDeviceId));
    CUDA_CHECK(hipMemPrefetchAsync(x, N / 4 * sizeof(char4), hipCpuDeviceId));
    CUDA_CHECK(hipMemPrefetchAsync(y, N * sizeof(float), hipCpuDeviceId));
    // TODO

    float sum_y = 0.0f;
    float sum_z = 0.0f;
    for (int i = 0; i < N; i++)
    {
      sum_y += y[i];
      sum_z += z[i];
    }
    std::cout << "CPU result: y[0] = " << y[0] << ", sum(y) = " << sum_y << ", z[0] = " << z[0] << ", sum(z) = " << sum_z << std::endl;

    printf("Starting CPU benchmark\n");

    auto start = std::chrono::high_resolution_clock::now();
    for (uint32_t i = 0; i < num_iterations; i++)
    {
      mul_cpu(N, A, x, y);
      post_mul_scale_cpu(N, y, A_scale, x_scale);
    }
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed = end - start;
    printf("%f ms per iteration\n", elapsed.count() / num_iterations * 1e3f);
  }

  CUDA_CHECK(hipFree(A));
  CUDA_CHECK(hipFree(x));
  CUDA_CHECK(hipFree(y));

  return 0;
}
#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include <stdio.h>
#include <chrono>
#include <cassert>

#define CUDA_CHECK(err)                                                         \
  do                                                                            \
  {                                                                             \
    hipError_t err_ = (err);                                                   \
    if (err_ != hipSuccess)                                                    \
    {                                                                           \
      fprintf(stderr, "CUDA error %d at %s:%d: %s\n", err_, __FILE__, __LINE__, \
              hipGetErrorString(err_));                                        \
      exit(1);                                                                  \
    }                                                                           \
  } while (0)

__global__ void
init_cuda(int n, float *A, float *x, float *y)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < n; i += stride)
  {
    for (int j = 0; j < n; j++)
    {
      A[i * n + j] = float(i % 5) - float(j % 7) * 0.3f;
    }
    x[i] = float(i % 3 - 1.2f);
    y[i] = 0.0f;
  }
}

void init_cpu(int n, float *A, float *x, float *y)
{
  for (int i = 0; i < n; i++)
  {
    for (int j = 0; j < n; j++)
    {
      A[i * n + j] = float(i % 5) - float(j % 7) * 0.3f;
    }
    x[i] = float(i % 3 - 1.2f);
    y[i] = 0.0f;
  }
}

__global__ void mul_gpu(int n, float const *__restrict__ A, float const *__restrict__ x, float *__restrict__ y)
{
  __shared__ float sdata[16][16];

  int i = blockIdx.y * blockDim.y + threadIdx.y;
  int j = blockIdx.x * (blockDim.x * 2) + threadIdx.x;

  if (i < n && j < n)
  {
    sdata[threadIdx.y][threadIdx.x] = A[i * n + j] * x[j] + A[i * n + j + blockDim.x] * x[j + blockDim.x];
    __syncthreads();

    for (int s = blockDim.x / 2; s > 0; s >>= 1)
    {
      if (threadIdx.x < s)
      {
        sdata[threadIdx.y][threadIdx.x] += sdata[threadIdx.y][threadIdx.x + s];
      }
      __syncthreads();
    }
    if (threadIdx.x == 0)
    {
      atomicAdd(&y[i], sdata[threadIdx.y][0]);
    }
  }
}

void mul_cpu(int n, float *A, float *x, float *y)
{
  for (int i = 0; i < n; i++)
  {
    float sum = 0.0f;
    for (int j = 0; j < n; j++)
    {
      sum += A[i * n + j] * x[j];
    }
    y[i] = sum;
  }
}

int main(void)
{
  int N = 4096;
  uint32_t num_iterations = 10;

  // GPU version
  {
    float *A, *x, *y;
    CUDA_CHECK(hipMallocManaged(&A, N * N * sizeof(float)));
    CUDA_CHECK(hipMallocManaged(&x, N * sizeof(float)));
    CUDA_CHECK(hipMallocManaged(&y, N * sizeof(float)));

    assert(N % 2 == 0);
    dim3 block_size(16, 16);
    dim3 grid_size((N / 2 + block_size.x - 1) / block_size.x, (N + block_size.y - 1) / block_size.y);

    init_cuda<<<grid_size, block_size>>>(N, A, x, y);
    mul_gpu<<<grid_size, block_size>>>(N, A, x, y);
    CUDA_CHECK(hipDeviceSynchronize());

    float sum_y = 0.0f;
    for (int i = 0; i < N; i++)
    {
      sum_y += y[i];
    }
    std::cout << "GPU result: y[0] = " << y[0] << ", sum(y) = " << sum_y << std::endl;

    CUDA_CHECK(hipMemPrefetchAsync(A, N * N * sizeof(float), 0));
    CUDA_CHECK(hipMemPrefetchAsync(x, N * sizeof(float), 0));
    CUDA_CHECK(hipMemPrefetchAsync(y, N * sizeof(float), 0));

    printf("Starting GPU benchmark\n");
    auto start = std::chrono::high_resolution_clock::now();
    for (uint32_t i = 0; i < num_iterations; i++)
    {
      mul_gpu<<<grid_size, block_size>>>(N, A, x, y);
      CUDA_CHECK(hipDeviceSynchronize());
    }
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed = end - start;
    printf("%f ms per iteration\n", elapsed.count() / num_iterations * 1e3f);

    CUDA_CHECK(hipFree(A));
    CUDA_CHECK(hipFree(x));
    CUDA_CHECK(hipFree(y));
  }

  // CPU version
  {
    float *A = (float *)malloc(N * N * sizeof(float));
    float *x = (float *)malloc(N * sizeof(float));
    float *y = (float *)malloc(N * sizeof(float));

    init_cpu(N, A, x, y);

    mul_cpu(N, A, x, y);

    float sum_y = 0.0f;
    for (int i = 0; i < N; i++)
    {
      sum_y += y[i];
    }
    std::cout << "CPU result: y[0] = " << y[0] << ", sum(y) = " << sum_y << std::endl;

    printf("Starting CPU benchmark\n");

    auto start = std::chrono::high_resolution_clock::now();
    for (uint32_t i = 0; i < num_iterations; i++)
    {
      mul_cpu(N, A, x, y);
    }
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed = end - start;
    printf("%f ms per iteration\n", elapsed.count() / num_iterations * 1e3f);

    free(A);
    free(x);
    free(y);
  }

  return 0;
}